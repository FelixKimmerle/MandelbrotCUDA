#include "hip/hip_runtime.h"
#include "Kernel.h"
#include <thrust/complex.h>

#define BLOCK_SIZE 16

__device__ uchar4 convert_one_pixel_to_rgb(float h, float s, float v)
{
    float r, g, b;

    float f = h / 60.0f;
    float hi = floorf(f);
    f = f - hi;
    float p = v * (1 - s);
    float q = v * (1 - s * f);
    float t = v * (1 - s * (1 - f));

    if (hi == 0.0f || hi == 6.0f)
    {
        r = v;
        g = t;
        b = p;
    }
    else if (hi == 1.0f)
    {
        r = q;
        g = v;
        b = p;
    }
    else if (hi == 2.0f)
    {
        r = p;
        g = v;
        b = t;
    }
    else if (hi == 3.0f)
    {
        r = p;
        g = q;
        b = v;
    }
    else if (hi == 4.0f)
    {
        r = t;
        g = p;
        b = v;
    }
    else
    {
        r = v;
        g = p;
        b = q;
    }

    unsigned char red = 255.0f * r;
    unsigned char green = 255.0f * g;
    unsigned char blue = 255.0f * b;
    unsigned char alpha = 255;
    return (uchar4){red, green, blue, alpha};
}

__device__ uchar4 GetColor(int n, int max, int color)
{
    if (color == 0)
    {
        double t = (double)n / (double)max;

        unsigned int r = (int)(9 * (1 - t) * t * t * t * 255);
        unsigned int g = (int)(15 * (1 - t) * (1 - t) * t * t * 255);
        unsigned int b = (int)(8.5 * (1 - t) * (1 - t) * (1 - t) * t * 255);

        return make_uchar4(r, g, b, 255);
    }
    else if (color == 1)
    {
        int N = 256;
        int N3 = N * N * N;
        double t = (double)n / (double)max;
        n = (int)(t * (double)N3);
        int b = n / (N * N);
        int nn = n - b * N * N;
        int r = nn / N;
        int g = nn - r * N;
        return make_uchar4(r, g, b, 255);
    }
    else if (color == 2)
    {
        float mmm = ((float)n / (float)max);
        return convert_one_pixel_to_rgb(mmm * 250, 1.0, 1.0);
    }
    else if (color == 3)
    {
        //float z = n%10*255/9;
        float z = ((n % 20) / 20.f) * 255;

        return convert_one_pixel_to_rgb(z, 1.0, 1.0);
    }
    else if (color == 4)
    {
        float z = n % 2 * 255;
        return make_uchar4(z, z, z, 255);
    }
}
__global__ void RenderKernel(uchar4 *dst, double SXSize, double SYSize, double FXSize, double FYSize, double FXMin, double FYMin, int iter_max, int smooth,double xpixelfact,double ypixelfact)
{
    int index = (threadIdx.x + blockIdx.x * blockDim.x);

    if (index < SXSize * SYSize)
    {

        int xc = index % (int)(SXSize);
        int yc = (int)((index - xc) / SXSize);

        unsigned int n = 0;
        double zx, zy, zx2, zy2;
        zx = zy = zx2 = zy2 = 0;
        double x = (double)xc * xpixelfact + FXMin; // x1
        double y = (double)yc * ypixelfact + FYMin; // x2

        for (; n < iter_max && zx2 + zy2 < 4; n++)
        {
            zy = 2 * zx * zy + y;
            zx = zx2 - zy2 + x;
            zx2 = zx * zx;
            zy2 = zy * zy;
        }
        
        dst[index] = GetColor(n, iter_max, smooth);        
    }
}

uchar4 *g_dstBuffer = NULL;
size_t g_BufferSize = 0;

void Render(hipGraphicsResource_t &dst, Dimention<double> &screen, Dimention<double> &fract, int iter_max, int smooth)
{
    hipGraphicsResource_t resources[1] = {dst};
    hipGraphicsMapResources(1, resources);
    hipArray *dstArray;
    hipGraphicsSubResourceGetMappedArray(&dstArray, dst, 0, 0);

    size_t bufferSize = screen.width() * screen.height() * sizeof(uchar4);
    if (g_BufferSize != bufferSize)
    {
        if (g_dstBuffer != NULL)
        {
            hipFree(g_dstBuffer);
        }
        g_BufferSize = bufferSize;
        hipMalloc(&g_dstBuffer, g_BufferSize);
    }

    size_t blocksW = (size_t)ceilf(screen.width() / (float)BLOCK_SIZE);
    size_t blocksH = (size_t)ceilf(screen.height() / (float)BLOCK_SIZE);
    int n = screen.width() * screen.height();

    double xpixelfact =  1.f/(double)screen.width() * fract.width();
    double ypixelfact = 1.f/(double)screen.height() * fract.height();
    RenderKernel<<<n / BLOCK_SIZE, BLOCK_SIZE>>>(g_dstBuffer, screen.width(), screen.height(), fract.width(), fract.height(), fract.x_min(), fract.y_min(), iter_max, smooth,xpixelfact,ypixelfact);

    //hipMemcpyToArray(dstArray, 0, 0, g_dstBuffer, bufferSize, hipMemcpyDeviceToDevice);
    hipMemcpy2DToArray(dstArray, 0, 0, g_dstBuffer, screen.width() * sizeof(uchar4), screen.width() * sizeof(uchar4), screen.height(), hipMemcpyDeviceToDevice);
    //TODO how to replace with hipMemcpy2DToArray() ?;
    hipGraphicsUnmapResources(1, resources);
}

__global__ void RenderKernelA(uchar4 *dst, double SXSize, double SYSize, double FXSize, double FYSize, double FXMin, double FYMin, int iter_max, int smooth, double radius, unsigned int size, thrust::complex<double> *xxx)
{
    int index = (threadIdx.x + blockIdx.x * blockDim.x);

    if (index < SXSize * SYSize)
    {

        int xc = index % (int)(SXSize);
        int yc = (int)((index - xc) / SXSize);

        double x = (double)xc / (double)SXSize * FXSize + FXMin; // x1
        double y = (double)yc / (double)SYSize * FYSize + FYMin; // x2

        int window_radius = (SXSize < SYSize) ? SXSize : SYSize;
        // find the complex number at the center of this pixel
        //thrust::complex<double> d0(radius * (2.f * xc - (double)SXSize) / (double)window_radius,
        // -radius * (2.f * yc - (double)SYSize) / (double)window_radius);

        thrust::complex<double> d0(x,y);

        int iter = 0;

        double zn_size;
        // run the iteration loop
        thrust::complex<double> dn = d0;
        do
        {
            dn *= xxx[iter] + dn;
            dn += d0;
            ++iter;
            zn_size = thrust::norm(xxx[iter] * thrust::complex<double>(0.5, 0.0) + dn);

            // use bailout radius of 256 for smooth coloring.
        } while (zn_size < 256 && iter < size);

        dst[index] = GetColor(iter, iter_max, smooth);
    }
}

void RenderA(hipGraphicsResource_t &dst, Dimention<double> &screen, Dimention<double> &fract, int iter_max, int smooth, double radius, unsigned int size, thrust::complex<double> *x)
{
    hipGraphicsResource_t resources[1] = {dst};
    hipGraphicsMapResources(1, resources);
    hipArray *dstArray;
    hipGraphicsSubResourceGetMappedArray(&dstArray, dst, 0, 0);
    thrust::complex<double> *xgpu;
    hipMalloc(&xgpu, size * sizeof(thrust::complex<double>));
    hipMemcpy(x, xgpu, size * sizeof(thrust::complex<double>), hipMemcpyHostToDevice);

    size_t bufferSize = screen.width() * screen.height() * sizeof(uchar4);
    if (g_BufferSize != bufferSize)
    {
        if (g_dstBuffer != NULL)
        {
            hipFree(g_dstBuffer);
        }
        g_BufferSize = bufferSize;
        hipMalloc(&g_dstBuffer, g_BufferSize);
    }

    size_t blocksW = (size_t)ceilf(screen.width() / (float)BLOCK_SIZE);
    size_t blocksH = (size_t)ceilf(screen.height() / (float)BLOCK_SIZE);
    int n = screen.width() * screen.height();
    RenderKernelA<<<n / BLOCK_SIZE, BLOCK_SIZE>>>(g_dstBuffer, screen.width(), screen.height(), fract.width(), fract.height(), fract.x_min(), fract.y_min(), iter_max, smooth, radius, size, xgpu);

    hipMemcpy2DToArray(dstArray, 0, 0, g_dstBuffer, screen.width() * sizeof(uchar4), screen.width() * sizeof(uchar4), screen.height(), hipMemcpyDeviceToDevice);
    //TODO how to replace with hipMemcpy2DToArray() ?;
    hipGraphicsUnmapResources(1, resources);
    hipFree(xgpu);
}